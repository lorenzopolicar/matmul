#include "hip/hip_runtime.h"
#include <matrixMultiplyGPU.cuh>
#define STUDENTID 46983820 //DO NOT REMOVE
/**
* @brief Implements an NxN matrix multiply C=A*B
*  		   	 	 	 			  		  		 	      
* @param[in] N : dimension of square matrix (NxN)
* @param[in] A : pointer to input NxN matrix
* @param[in] B : pointer to input NxN matrix
* @param[out] C : pointer to output NxN matrix
* @param[in] flags : pointer to array of integers which can be used for debugging and performance tweaks. Optional. If unused, set to zero
* @param[in] flagCount : the length of the flags array
* @return : your student ID
*  		   	 	 	 			  		  		 	      
* */
__host__ int matrixMultiply_GPU(int N, const floatTypeCUDA* A, const floatTypeCUDA* B, floatTypeCUDA* C, int* flags, int flagCount){  		   	 	 	 			  		  		 	      
if (N<=0) { return STUDENTID;}//Your code must be able to deal with N=0 scenario without crashing.  		   	 	 	 			  		  		 	      

//WRITE YOUR CODE HERE

return STUDENTID;  		   	 	 	 			  		  		 	      

}  		   	 	 	 			  		  		 	      

//The kernel (device code) parameters have been setup almost the same as the host code, except the flags are passed in individually rather than as a pointer. This is done just so you don't have to copy the parameters to GPU memory first, you'll be able to pass in up to 3 on the function call.  		   	 	 	 			  		  		 	      
__global__ void matrixMultiplyKernel_GPU(int N, const floatTypeCUDA* A, const floatTypeCUDA* B, floatTypeCUDA* C, int flag0, int flag1, int flag2){  		   	 	 	 			  		  		 	      

}  		   	 	 	 			  		  		 	      
